#include "hip/hip_runtime.h"
/*
  cuda backend for mandelbrot computing
  zeFresk
*/

#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/complex.h>

#include "types.hpp"

// give f_n(c) where f is mandelbrot function and c is a complex point
struct zn_functor {
	const size_t iterations;
	zn_functor(size_t iters) : iterations{iters} {}

	__device__
	float operator()(complex_t& z, complex_t const& c) {
		for (size_t i = 0; i < iterations; ++i) {
			if (thrust::abs(z) > static_cast<real_t>(2.f))
				return static_cast<float>(i) / static_cast<float>(iterations);
			z *= z;
			z += c;
		}
		return 1.f;
	}
};

std::vector<float> cuda_compute_normalize_all(std::vector<complex_t> const& vec, size_t iterations) {
	thrust::device_vector<complex_t> c(vec.size());
	thrust::copy(vec.begin(), vec.end(), c.begin());
	thrust::device_vector<complex_t> zn = c; // 0+0i +c array,  z1 not z0 !
	thrust::device_vector<float> normalized(vec.size());

	// z*z + c iterations times + normalize
	thrust::transform(zn.begin(), zn.end(), c.begin(), normalized.begin(), zn_functor(iterations-1));
	
	// copy normalized values
	std::vector<float> ret(vec.size());
	thrust::host_vector<float> hret = normalized;
	thrust::copy(hret.begin(), hret.end(), ret.begin());
	return ret;
}